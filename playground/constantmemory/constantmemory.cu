#include "hip/hip_runtime.h"
#include"spqlios++.hpp"
#include <random>
#include <cassert>
#include <iostream>
#include <cmath>

using namespace TFHEpp;
using namespace std;

template<typename T = uint32_t,uint32_t N = DEF_N>
__global__ void cudaTwistMulInvert(double* res, T* a, const double* twist){
    unsigned int tid = threadIdx.x;
    unsigned int bdim = blockDim.x;
        for (int i = tid; i < N / 2; i+=bdim) {
            const double are = static_cast<double>(static_cast<typename make_signed<T>::type>(a[i]));
            const double aim = static_cast<double>(static_cast<typename make_signed<T>::type>(a[i+N/2]));
            const double aimbim = aim * twist[i + N / 2];
            const double arebim = are * twist[i + N / 2];
            res[i] = are * twist[i] - aimbim;
            res[i + N / 2] = aim * twist[i] + arebim;
        }
        __threadfence();
    }

int main( int argc, char** argv) 
{
    const array<double,DEF_N> h_twistlvl1 = SPQLIOSpp::TwistGen<DEF_N>();
    double* twistlvl1;
    hipMalloc( (void**) &twistlvl1, sizeof(h_twistlvl1));
    hipMemcpy( twistlvl1, h_twistlvl1.data(), sizeof(h_twistlvl1),hipMemcpyHostToDevice);

    random_device seed_gen;
    default_random_engine engine(seed_gen());
    uniform_int_distribution<uint32_t> Torus32dist(0, UINT32_MAX);

    Polynomiallvl1 a;
    for (uint32_t &i : a) i = Torus32dist(engine);

    PolynomialInFDlvl1 h_res,res;
    SPQLIOSpp::TwistMulInvert<uint32_t,DEF_N>(h_res,a,h_twistlvl1);

    uint32_t* d_a;
    double* d_res;
    hipMalloc( (void**) &d_a, sizeof(a));
    hipMalloc( (void**) &d_res, sizeof(res));
    hipMemcpy(d_a,a.data(),sizeof(a),hipMemcpyHostToDevice);
    cudaTwistMulInvert<<<1,16>>>(d_res,d_a,twistlvl1);
    hipMemcpy(res.data(),d_res,sizeof(res),hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    for(int i = 0;i<DEF_N;i++) assert(abs(res[i]-h_res[i])<1e-6);
    hipFree(d_a);
    hipFree(d_res);
    hipFree(twistlvl1);
    cout<<"PASS"<<endl;
}