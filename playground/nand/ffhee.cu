#include"ffhee.cuh"
#include"gatebootstrapping.cuuh"
#include"spculios/spculios.cuh"

namespace FFHEE{

__device__ cuBootStrappingKeyFFTlvl01 d_bkfftlvl01;
__device__ cuKeySwitchingKey d_ksk;
__device__ cuTLWElvl0 d_ca, d_cb, d_res;

void FFHEEinit(TFHEpp::GateKey &gk){
    FFTinit();
    hipMemcpyToSymbol(HIP_SYMBOL(d_bkfftlvl01),gk.bkfftlvl01.data(),sizeof(gk.bkfftlvl01));
    hipMemcpyToSymbol(HIP_SYMBOL(d_ksk),gk.ksk.data(),sizeof(gk.ksk));
}

void cuHomNAND(TFHEpp::TLWElvl0 &res, const TFHEpp::TLWElvl0 &ca, const TFHEpp::TLWElvl0 &cb,
             const TFHEpp::GateKey &gk){
    hipMemcpyToSymbolAsync(HIP_SYMBOL(d_ca),ca.data(),sizeof(ca));
    hipMemcpyToSymbolAsync(HIP_SYMBOL(d_cb),ca.data(),sizeof(cb));
}
}