#include "hip/hip_runtime.h"
#include<params.hpp>
#include<mulfft.cuh>
#include <random>

using namespace TFHEpp;

void main(){
    const uint32_t num_test = 1000;
    random_device seed_gen;
    default_random_engine engine(seed_gen());
    uniform_int_distribution<uint32_t> Torus32dist(0, UINT32_MAX);

    cout << "Start LVL1 test." << endl;
    for (int test; test < num_test; test++) {
        Polynomiallvl1 a,res;
        for (uint32_t &i : a) i = Torus32dist(engine);
        FFHEE::FFTlvl1Test(res,a);
        for (int i = 0; i < DEF_N; i++)
            assert(abs(static_cast<int32_t>(a[i] - res[i])) <= 1);
    }
    cout << "FFT Passed" << endl
}